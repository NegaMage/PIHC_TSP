
#include <hip/hip_runtime.h>
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>
#include <assert.h>

/* Euclidean distance calculation */
__host__ __device__ long distD(int i,int j,float *x,float*y)
{
	float dx=x[i]-x[j];
	float dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}
//all these strats are for the two opt move,
/*A kenel function that finds a minimal weighted neighbor using TPR mapping strategy*/
__global__ void tsp_tpr(float *pox,float *poy,long initcost,unsigned long long *dst_tid,long cit)
{
	//threads per row strategy
	long id,j;
	register long change,mincost=initcost,cost;
	long i=threadIdx.x+blockIdx.x*blockDim.x;
	if(i < cit)
	{	//
		
		for(j=i+1;j<cit;j++)
		{//pox and poy are arrays that store the positions (x y) of ith city
			change = 0; cost=initcost;
			change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
			cost+=change;	
			if(cost < mincost)
			{
				mincost = cost;
				id = i * (cit-1)+(j-1)-i*(i+1)/2;	
			}	 

		}
		if(mincost < initcost)
			 atomicMin(dst_tid, ((unsigned long long)mincost << 32) | id);

	}
	
}

/*A kenel function that finds a minimal weighted neighbor using TPRED mapping strategy*/
__global__ void tsp_tpred(float *pox,float *poy,long initcost,unsigned long long *dst_tid,long cit,long itr)
{
	long id,j,k;
	register long change,mincost=initcost,cost;
	long i=threadIdx.x+blockIdx.x*blockDim.x;
	if(i < cit)
	{
		//itr is how many iterations we can stand to do.
		for(k=0;k<itr;k++)
		{
			change = 0; cost=initcost;
			j=(i+1+k)%cit;
			change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
			cost+=change;	
			if(cost < mincost)
			{
				mincost = cost;
				if(i < j)
					id = i * (cit-1)+(j-1)-i*(i+1)/2;	
				else
					id = j * (cit-1)+(i-1)-j*(j+1)/2;	

			}	 

		}
		if(mincost < initcost)
			 atomicMin(dst_tid, ((unsigned long long)mincost << 32) | id);
	}
}

/*A kenel function that finds a minimal weighted neighbor using TPRC mapping strategy*/
__global__ void tsp_tprc(float *pox,float *poy,long initcost,unsigned long long *dst_tid,long cit)
{

	long id;
	long change,cost;
	long i=threadIdx.x+blockIdx.x*blockDim.x;
	long j=threadIdx.y+blockIdx.y*blockDim.y;
	//if city in bounds and the column you choose is more than the row, so there is no repeat issues
	if(i < cit && j < cit && i < j)
	{
		
			change = 0; cost = initcost;
			change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
			cost+=change;	
			if(change < 0)
			{
				id = i * (cit - 1) + (j - 1) - i * (i + 1) / 2;	
				atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
			}	 

	}
	
}

/*A kenel function that finds a minimal weighted neighbor using TPN mapping strategy*/
__global__ void tsp_tpn(float *pox,float *poy,long cost,unsigned long long *dst_tid,long cit,long sol)
{

	long i,j;
	register long change=0;
	int id=threadIdx.x+blockIdx.x*blockDim.x;
	if(id<sol)
	{
		
		i=cit-2-floorf(((int)__dsqrt_rn(8*(sol-id-1)+1)-1)/2);
		j=id-i*(cit-1)+(i*(i+1)/2)+1;
		change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
		cost+=change;	
		if(change < 0)
			 atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
		
	}
	
}

/* At each IHC steps, XY coordinates are arranged using next initial solution's order*/
void twoOpt(long x,long y,float *pox,float *poy)
{
	float *tmp_x,*tmp_y;
	int i,j;
	
	tmp_x=(float*)malloc(sizeof(float)*(y-x));	
	tmp_y=(float*)malloc(sizeof(float)*(y-x));	
	for(j=0,i=y;i>x;i--,j++)
	{
		tmp_x[j]=pox[i];
		tmp_y[j]=poy[i];
	}
	for(j=0,i=x+1;i<=y;i++,j++)
	{
		pox[i]=tmp_x[j];
		poy[i]=tmp_y[j];
	}
	free(tmp_x);
	free(tmp_y);

}


/*Arranges XY coordinates in initial solution's order*/
void setCoord(int *r,float *posx,float *posy,float *px,float *py,long cities)
{
	for(int i=0;i<cities;i++)
	{
		px[i]=posx[r[i]];
		py[i]=posy[r[i]];
	}
}
/* Initial solution construction using NN */
long nn_init(int *route,long cities,float *posx,float*posy)
{	//route stores the route taken, cities is the number of cities, posx and posy are the positions of the ith city
	route[0]=0;
	int k=1,i=0,j;
	float min;
	int minj,mini,count=1,flag=0;
	long dst=0;
	int *visited=(int*)calloc(cities,sizeof(int));
	visited[0]=1;
	while(count!=cities)
	{
		flag=0;
		for(j=1;j<cities;j++)
		{	//if j isn't visited yet
			if(i!=j && !visited[j])
			{
				min=distD(i,j,posx,posy);
				minj=j;
				break;	
			}
		}
		//for the minimum cost j
		for(j=minj+1;j<cities;j++)
		{//for every node from the min cost, if you haven't visited, then check and generate the pair
			
			 if( !visited[j])
			{
				if(min>distD(i,j,posx,posy))
				{
					min=distD(i,j,posx,posy);
					mini=j;
					flag=1;				
				}
			}
		}
		if(flag==0)
			i=minj;
		else
			i=mini;
		dst+=min;
		route[k++]=i;
		visited[i]=1;
		count++;
	}
	free(visited);
	dst+=distD(route[0],route[cities-1],posx,posy);
	return dst;
}
/* Initial solution construction using sequenced approach */
void seq_init(int*route,long N)
{
	int i;
	for(i=0;i<N;i++)
		route[i]=i;
}

/* Initial solution construction using random approach */
void random_init(int *route,long cities)
{
	int i=0,j;
	int *visited = (int*)calloc(cities,sizeof(int));
	
	while(i<cities)
	{
		//srand (clock() );
		j=rand() % cities;
		if(visited[j])
		{
			continue;
		}
		else
		{
			route[i]=j;
			visited[j]=1;
			i++;	
		}

	}
	free(visited);
}

struct nearest_insert
{
	int city;
	struct nearest_insert *next;
};
struct odd_degree
{
	int city;
	struct odd_degree *next;
};	
struct rev_visit
{
int i,j;
struct rev_visit *next;
};

/* Initial solution construction using nearest insertion approach */
void nearest_insertion(int *r, float *posx, float *posy, long cities)
{
	struct nearest_insert *node,*p1,*tmp,*current,*route,*first = NULL;
	int i,j,dist,min=0;
	int count,minI,minJ; 
	int min_diff,diff,min_i,min_j; 
	int *v;
	v = (int *)calloc(cities, sizeof(int));
	node = (struct nearest_insert *)malloc(sizeof(struct nearest_insert ));
	node->city = 0;
	node->next = NULL;
	first = node;
	current = node;
	count = 1;
	v[0]=1;
	while(count != cities)
	{//as long as there are new cities
		min = 0;
		for(route = first; route != NULL; route=route->next)
		{//traverse the whole route to find the shortest edge
			i = route->city;
			for(j = 0; j < cities; j++)
			{
				if(i !=j &&!v[j])
				{
					dist = distD(i,j,posx,posy);
					if(min==0)
					{
						min=dist;
						minI=i;
						minJ=j;
	
					}
					if(min>dist)
					{
						min=dist;
						minI=i;
						minJ=j;
					}
				}
			}
		}
		//setting that node on edge to explored
		v[minJ]=1;
		//when you're starting out
		if(count < 3)
		{
			if(first->city == minI)
			{
				if(first->next == NULL)
				{
					node = (struct nearest_insert *)malloc(sizeof(struct nearest_insert ));
					node->city = minJ;
					node->next = NULL;
					first->next = node;
					current = current->next;
				}
				else
				{
					tmp = first->next;
					node = (struct nearest_insert *)malloc(sizeof(struct nearest_insert ));
					node->city = minJ;
					node->next = tmp;
					first->next = node;
				}
			}
			else if(current->city == minI)
			{
					node = (struct nearest_insert *)malloc(sizeof(struct nearest_insert ));
					node->city = minJ;
					node->next = NULL;
					current->next = node;
					current = current->next;
			}
			else
			{
				p1 = first->next;
				while (p1->city != minI)
					p1=p1->next;
				tmp = p1->next;
				node = (struct nearest_insert *)malloc(sizeof(struct nearest_insert ));
				node->city = minJ;
				node->next = tmp;
				p1->next = node;
			}
		}
		else
		{//more than 3 nodes
			p1 = first;
			min_i = p1->city;
			min_j = p1->next->city;
			min_diff = distD(min_i,minJ,posx,posy) + distD(minJ,min_j,posx,posy) - distD(min_i,min_j,posx,posy);
			p1 = p1->next;
			while(p1->next!=NULL)
			{//go through the path
				i = p1->city;
				j = p1->next->city;
				//check two opt
				diff = distD(i,minJ,posx,posy) + distD(minJ,j,posx,posy) - distD(i,j,posx,posy);
				if(min_diff > diff )
				{
					min_diff = diff;
					min_i = i;
					min_j = j;
				}
				p1 = p1->next;	
			}
			i = p1->city;
			j = 0;
			diff = distD(i,minJ,posx,posy) + distD(minJ,j,posx,posy) - distD(i,j,posx,posy);
			//and cycle around
			if(min_diff > diff )
			{
				min_diff = diff;
				min_i = i;
				min_j = j;
			}

			if(current->city == min_i)
			{
				node = (struct nearest_insert *)malloc(sizeof(struct nearest_insert ));
				node->city = minJ;
				node->next = NULL;
				current->next = node;
				current = current->next;
			}
			else
			{
				p1 = first;
				while (p1->city != min_i)
				{	p1=p1->next;}
				tmp = p1->next;
				node = (struct nearest_insert *)malloc(sizeof(struct nearest_insert ));
				node->city = minJ;
				node->next = tmp;
				p1->next = node;
			}
		}
		count++;
	}
	i=0;
	p1=first;
	while(p1!=NULL)
	{
		r[i] = p1->city;
		p1=p1->next;
		i++;
	}
}

struct greedy
{
	int city;
	struct greedy *next;
};
/* Initial solution construction using greedy approach */
void greedy(int *r, float *posx, float *posy, long cities)
{
	struct greedy *node,*p1,*current,*first = NULL;

	int i,j,min=0,dist;
	int count,minI,minJ; 
	int *v;

	v = (int *)calloc(cities, sizeof(int));
	node = (struct greedy *)malloc(sizeof(struct greedy ));

	node->city = 0;
	node->next = NULL;
	first = node;
	current = node;

	count = 1;
	v[0]=1;
	min = 0;

	while(count != cities)
	{	//operate from the first city,
		i = first->city;
		min = 0;
		//keep exploring cities
		for(j = 0; j < cities; j++)
		{	//until you find a new one
			if(!v[j] && i != j)
			{	//calc distance and store min dist
				dist = distD(i,j,posx,posy);
				if(min==0)
				{
					min=dist;
					minI=i;
					minJ=j;

				}
				if(min>dist)
				{
					min=dist;
					minI=i;
					minJ=j;
				}
			}
		}
		//if this is not the first pass
		if(first != current)
		{
			i = current->city;
			for(j = 0; j < cities; j++)
			{//then store into list. basically the same because we needed cases
				if(!v[j] && i != j)
				{
					dist = distD(i,j,posx,posy);
					if(min>dist)
					{
						min=dist;
						minI=i;
						minJ=j;
					}
				}
			}
		}
		v[minJ]=1;
		
		if(first->city == minI)
		{
			if(first->next == NULL)
			{
				node = (struct greedy *)malloc(sizeof(struct greedy ));
				node->city = minJ;
				node->next = NULL;
				first->next = node;
				current = current->next;
			}
			else
			{
				node = (struct greedy *)malloc(sizeof(struct greedy ));
				node->city = minJ;
				node->next = first;
				first = node;
			}
		}
		else
		{
			if (current->city == minI)
			{
				node = (struct greedy *)malloc(sizeof(struct greedy ));
				node->city = minJ;
				node->next = NULL;
				current->next = node;
				current = current->next;
			}
		}
	count++;
	}

	i=0;
	p1=first;
	while(p1!=NULL)
	{
		r[i] = p1->city;
		p1=p1->next;
		i++;
	}

}
struct visit_list
{
	int city;
	struct visit_list *next;
};
struct MST
{
	int i,j,weight;
	struct MST *next;
	struct MST *prev;
};
struct eul_tour
{
	int city;
	struct eul_tour *next;	
	struct eul_tour *prev;	
};
/* Initial solution construction using MST approach */
//minimum spanning tree
void mst_init(int *r, float *posx, float *posy, long cities)
{
	int *deg,*var_deg,dist;
	int i,j, min,min_i,min_j, count,*v;

	struct eul_tour *et,*top=NULL,*curr, *node1,*rev;
	struct visit_list *first=NULL,*current, *p1,*visited;
	struct MST *head =NULL, *cur, *node,*p;

	deg = (int*)calloc(cities,sizeof(int) );
	v = (int*) calloc(cities, sizeof(int));
	visited = (struct visit_list*)malloc(sizeof(struct visit_list));

	visited->city = 0;
	visited->next = NULL;
	first = visited;
	current = first;
	count = 1;
	p1 =first;
	v[0] = 1;
	while(count != cities )
	{	//while all cities aren't explored
		min = 0;
	
		for(p1 = first; p1!=NULL; p1=p1->next)
		{	
			//fix a node,
			i = p1->city;
			for(j = 0; j < cities; j++)
			{
				//check and find the smallest edge with that node
				if(i != j && !v[j])
				{
					dist = distD(i,j,posx,posy);
					if(min == 0 )
					{
						min = dist;
						min_i =i;
						min_j =j;

					}
					if(min > dist)
					{
						min = dist;
						min_i =i;
						min_j =j;
					}
				}
		
			}
		
		}
		v[min_j] =1;
		visited = (struct visit_list*)malloc(sizeof(struct visit_list));
		visited->city = min_j;
		visited->next = NULL;
		current->next =visited;
		current = current->next;
		//and now add that edge
		deg[min_i]+=1;
		deg[min_j]+=1;
		//make a node of the mst,
		//and add that edge
		node = (struct MST*)malloc(sizeof(struct MST));
		node->i = min_i;
		node->j = min_j;
		node->weight = min;
		node->next = NULL;
		//linked list stuff
		if(head == NULL)
		{
			node->prev = NULL;
			head = node;
			cur = node;
		}	
		else
		{
			node->prev = cur;
			cur->next = node;
			cur = cur->next;
		}
	count++; 
	}

	v = (int*) calloc(cities, sizeof(int));
	var_deg = (int*) calloc(cities, sizeof(int));
	p = head;
	//find a leaf,
	while(deg[p->i] != 1 && deg[p->j] != 1)
		p = p->next;
	//take the leaf city, 
	if(deg[p->i] == 1 )
	{	//take the leaf,make it a node in the euler tour, make the jth node the other node of the edge in the tour,
		i = p->i;
		node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
		node1->city = i;
		node1->next = NULL;
		node1->prev = NULL;
		top = node1;
		curr = node1;
		v[i] = 1;
		var_deg[i]++;
		node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
		node1->city = p->j;
		node1->next = NULL;
		node1->prev = curr;
		curr->next = node1;
		curr = curr->next;
		j = p->j;
		v[j] = 1;	
		var_deg[j]++;
	}
	else
	{
		i = p->j;
		node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
		node1->city = i;
		node1->next = NULL;
		node1->prev = NULL;
		top = node1;
		curr = node1;
		v[i] = 1;
		var_deg[i]++;

		node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
		node1->city = p->i;
		node1->next = NULL;
		node1->prev = curr;
		curr->next = node1;
		curr = curr->next;
		j = p->i;
		v[j] = 1;	
		var_deg[j]++;
	}
	//now we have 2 nodes, ie one edge,
	count = 2;
	p = head;
	while(count != cities)
	{
		if(deg[j]!= 1)
		{
			if(p->i == j && !v[p->j])
			{
				node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
				node1->city = p->j;
				node1->next = NULL;
				node1->prev = curr;
				curr->next = node1;
				curr = curr->next;
				j = p->j;
				v[j] = 1;
				var_deg[p->i]++;	
				var_deg[p->j]++;	
				count++;
				p = p->next;
			}
			else if(p->j == j && !v[p->i])
			{
				node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
				node1->city = p->i;
				node1->next = NULL;
				node1->prev = curr;
				curr->next = node1;
				curr = curr->next;
				j = p->i;
				v[j] = 1;	
				var_deg[p->i]++;	
				var_deg[p->j]++;	
				count++;
				p = p->next;
			}
			else
			{
				p = head;
				while( (p->i != j || v[p->j]) && (p->j != j || v[p->i]) )
					p = p->next;
			}
		}
		else
		{
			rev = curr->prev;
			while(deg[rev->city] == var_deg[rev->city])
			{
				rev = rev ->prev;			
			}
			
			j = rev->city;
			p = head;
		}	 
	}

	v = (int*) calloc(cities, sizeof(int));
	i=0;
	et = top;
	while(et != NULL)
	{
		if(v[et->city] == 0)
		{
			r[i++] = et->city; 		
			v[et->city] = 1;		
		}
		et = et->next;	
	}

}
//if the edge exists, then return 1
int searchEdge(int min_i,int min_j, struct MST * p)
{
	int flag =0;
		while(p != NULL )
		{
			if( (p->i == min_i && p->j == min_j) || (p->i == min_j && p->j == min_i ) )
			{
				flag = 1;
				break; 
			}
			p = p->next;	
		}
	if(flag == 1 )
		return 1;
	else
		return 0;

}

/* Initial solution construction using Christofides' approach */
void christofide_init(int *r, float *posx, float *posy, long cities)
{
	int *deg,*var_deg,dist,flg=0;
	int i,j, min,min_i,min_j, count,*v,size;
	int *odd_array,flag = 0;
	FILE *fp; char line[100];

	struct eul_tour *et,*top=NULL,*curr, *node1,*rev;
	struct visit_list *first=NULL,*current, *p1,*visited;
	struct MST *head =NULL, *cur, *node,*p;
	struct odd_degree *init=NULL, *at, *odd;
	struct rev_visit* rev_node=NULL,*loop;

	deg = (int*)calloc(cities,sizeof(int));
	v = (int*) calloc(cities, sizeof(int));
	visited = (struct visit_list*)malloc(sizeof(struct visit_list));

	visited->city = 0;
	visited->next = NULL;
	first = visited;
	current = first;
	count = 1;
	p1 =first;
	v[0] = 1;

	while(count != cities )
	{	
		min = 0;
	
		for(p1 = first; p1!=NULL; p1=p1->next)
		{	

			i = p1->city;
			for(j = 0; j < cities; j++)
			{
				if(i != j && !v[j])
				{
					dist = distD(i,j,posx,posy);
					if(min == 0 )
					{
						min = dist;
						min_i =i;
						min_j =j;

					}
					if(min > dist)
					{
						min = dist;
						min_i =i;
						min_j =j;
					}
				}
		
			}

		}
		v[min_j] =1;
		visited = (struct visit_list*)malloc(sizeof(struct visit_list));
		visited->city = min_j;
		visited->next = NULL;
		current->next =visited;
		current = current->next;
	
		deg[min_i]+=1;
		deg[min_j]+=1;

		node = (struct MST*)malloc(sizeof(struct MST));
		node->i = min_i;
		node->j = min_j;
		node->weight = min;
		node->next = NULL;

		if(head == NULL)
		{
			node->prev = NULL;
			head = node;
			cur = node;
		}	
		else
		{
			node->prev = cur;
			cur->next = node;
			cur = cur->next;
		}
	count++; 
	}
	p = head;
	size = 0;
	//make set of all odd degree nodes,
	for(i = 0; i < cities; i++)
	{
		if(deg[i]%2 != 0)
		{
	
			odd = (struct odd_degree*)malloc(sizeof(struct odd_degree));
			odd->city = i;
			odd->next = NULL;
			if(init == NULL)
			{
				init = odd;
				at = odd;
			}
			else
			{
				at->next = odd;
				at = at->next;

			}
		size++;
		}
	}
	
	odd_array = (int*)malloc(sizeof(int)*size);
	odd = init;
	i = 0;

	while(odd != NULL)
	{
		odd_array[i++] = odd->city;
		odd = odd->next;
	}
	//odd_array has all nodes with odd degrees
	v = (int*) calloc(size, sizeof(int));
	assert(size % 2 == 0);
	fp = fopen("odd_edges.txt", "w");
	//foul play case
	assert(size >= 2);
	fprintf(fp, "%d %d\n", size, (size*(size-1))/2);
	for (i = 0; i < size; i++) 
	{
		for (j = i+1; j < size; j++) 
		{
		fprintf(fp, "%d %d %ld\n", i, j, distD(odd_array[i],odd_array[j],posx,posy));
		}
	}
	fclose(fp);
	if(system("/home/sparklab/pramod/blossom5-v2.05.src/blossom5 -e odd_edges.txt -w min_edges.txt") != 0) 
	{
		printf("\nError: please install blossom5 matching code\n");
		exit(-1);
	}

	fp = fopen("min_edges.txt", "r");
	fgets(line, sizeof(line), fp); 
	for (i = 0; i < size/2; i++) 
	{
		assert(fgets(line, sizeof(line), fp) != NULL); 
		assert(sscanf(line, "%d %d", &i, &j) == 2); 
		if(searchEdge(odd_array[i],odd_array[j],head) ==  0)
		{
			deg[odd_array[i]]+=1;
			deg[odd_array[j]]+=1;

			node = (struct MST*)malloc(sizeof(struct MST));
			node->i = odd_array[i];
			node->j = odd_array[j];
			node->weight = distD(odd_array[i],odd_array[j], posx, posy);
			node->next = NULL;
			node->prev = cur;
			cur->next = node;
			cur = cur->next;
		}
	}
	fclose(fp); 

	v = (int*) calloc(cities, sizeof(int));
	var_deg = (int*) calloc(cities, sizeof(int));
	p = head;
	while(p != NULL)
	{
		if(deg[p->i] != 1 && deg[p->j] != 1)
		{	
			p = p->next;
		}
		else
		{
			flag = 1;
			break;		
		}
		
	}
	if(flag == 1)
	{
		if(deg[p->i] == 1 )
		{	i = p->i;
			node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
			node1->city = i;
			node1->next = NULL;
			node1->prev = NULL;
			top = node1;
			curr = node1;
			v[i] = 1;
			var_deg[i]++;

			node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
			node1->city = p->j;
			node1->next = NULL;
			node1->prev = curr;
			curr->next = node1;
			curr = curr->next;

			j = p->j;
			v[j] = 1;	
			var_deg[j]++;
		}
		else
		{
			i = p->j;
			node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
			node1->city = i;
			node1->next = NULL;
			node1->prev = NULL;
			top = node1;
			curr = node1;
			v[i] = 1;
			var_deg[i]++;

			node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
			node1->city = p->i;
			node1->next = NULL;
			node1->prev = curr;
			curr->next = node1;
			curr = curr->next;

			j = p->i;
			v[j] = 1;	
			var_deg[j]++;
		}
		count = 2;
		p = head;
		while(count != cities)
		{
			if(deg[j]!= 1)
			{
				if(p->i == j && !v[p->j])
				{
					node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
					node1->city = p->j;
					node1->next = NULL;
					node1->prev = curr;
					curr->next = node1;
					curr = curr->next;
					
					var_deg[p->i]++;	
					var_deg[p->j]++;	
					count++;
					
					j = p->j;
					v[j] = 1;
					p = p->next;
					if(p == NULL)
						p = head;
					
				}
				else if(p->j == j && !v[p->i])
				{
					node1 = (struct eul_tour*)malloc(sizeof(struct eul_tour));
					node1->city = p->i;
					node1->next = NULL;
					node1->prev = curr;
					curr->next = node1;
					curr = curr->next;

					var_deg[p->i]++;	
					var_deg[p->j]++;	
					count++;
			
					j = p->i;
					v[j] = 1;	
					p = p->next;
					if(p == NULL)
						p = head;
					

				}
				else
				{
					p = head;
					while(p != NULL)
					{
						if( (p->i != j || v[p->j]) && (p->j != j || v[p->i]) )
						{
							p = p->next;

						}
						else
						{
							flg = 1;
							break;
						}
					}
					if(flg == 0)
					{
						var_deg[j]++;
						et = curr-> prev;
						if(rev_node == NULL)
						{
							loop = (struct rev_visit *)malloc(sizeof(struct rev_visit));
							loop->i = j;
							while(deg[et->city] == var_deg[et->city] || et->city == j)
							{
								et = et-> prev;
							}
							j = et->city;
							loop->j = j;
							rev_node = loop;
							p = head;
						}
						else
						{
							if(j == rev_node->i || j == rev_node->j)
							{
								i = j == rev_node->i ? rev_node->j :rev_node->i;
								while(deg[et->city]==var_deg[et->city]|| et->city == j || et->city == i)
								{
									et = et-> prev;
								}
							}
							else
							{
								while(deg[et->city] == var_deg[et->city] || et->city == j)
								{
									et = et-> prev;
								}
							}
							rev_node->i = j;
							j = et->city;
							rev_node->j = j;
							p = head;
						}
						
					}
					
					flg = 0;	
				}
			}
			else
			{
				rev = curr->prev;
				while(deg[rev->city] == var_deg[rev->city] || rev->city == j)
				{
					rev = rev ->prev;			
				}
				j = rev->city;
				p = head;
			}	 
		}

		v = (int*) calloc(cities, sizeof(int));
		i=0;
		et = top;
		while(et != NULL)
		{
			if(v[et->city] == 0)
			{
				r[i++] = et->city; 		
				v[et->city] = 1;		
			}
			et = et->next;	
		}
	}
	else
	{
		v = (int*) calloc(cities, sizeof(int));
		p = head;
		i = 0;
		while(i != cities )
		{
			if(v[p->i] == 0)
			{
				r[i++] = p->i; 		
				v[p->i] = 1;		
			}
			if(v[p->j] == 0)
			{
				r[i++] = p->j; 		
				v[p->j] = 1;		
			}
			
			p = p->next;	
		}
	}
}

/* Initial solution construction using Clarke-Wright approach */
struct init_route
{
	int city;
	struct init_route *next;
};
struct clarke_wright
{
	int i,j, save;
	struct clarke_wright *next;
};

void clarke_wright_init(int *r, float *posx, float *posy, long cities, long no_pairs)
{
	int i,j,cnt;
	int *v;

	struct clarke_wright *cw,*cur,*cw1,*cw2;
	struct clarke_wright *top = NULL;

	for(i=1; i<cities-1; i++)
		for(j=i+1; j<cities; j++)
		{
			cw = (struct clarke_wright*)malloc(sizeof(struct clarke_wright) );
			cw->save = distD(0,i,posx,posy) + distD(0,j,posx,posy) - distD(i,j,posx,posy);
			cw->i = i;
			cw->j = j;		
			if(top==NULL)
			{
				cw->next= NULL;
				top = cw;			
				cur = cw;
			}
			else if( cw->save > top->save)
			{
			
				cw->next = top;
				top = cw;
			}
			else if (cw->save > cur->save && cw->save < top->save && cur != top)
			{
				cw1 = top;
				cw2 = top->next;
				while(cw2->save > cw->save)
				{
					cw2 = cw2->next;
					cw1 = cw1->next;

				}
				cw->next = cw2;
				cw1->next = cw;

			}
			else
			{
				cw->next = NULL;
				cur->next =cw;
				cur = cur->next;
			}
			
		}
	i = 0; 
	r[i++] = 0;	
	v=(int*)calloc(cities,sizeof(int));
	v[0] = 1;
	cw = top;
	r[i++] = cw->i;	
	v[cw->i] = 1;

	r[i++] = cw->j;	
	v[cw->j] = 1;
	cnt = 3;
	cw = cw->next;
	while(cnt != cities)
	{
		if( !v[cw->i] && !v[cw->j] )
		{
			r[i++] = cw->i;	
			v[cw->i] = 1;

			r[i++] = cw->j;	
			v[cw->j] = 1;
			cnt+=2;
		}
		else if( !v[cw->i]  )
		{
			r[i++] = cw->i;	
			v[cw->i] = 1;
			cnt++;
		}
		else if( !v[cw->j]  )
		{
			r[i++] = cw->j;	
			v[cw->j] = 1;
			cnt++;

		}
		cw = cw->next;

	}
}
void routeChecker(long N,int *r)
{
	int *v,i,flag=0;
	v=(int*)calloc(N,sizeof(int));	

	for(i=0;i<N;i++)
		v[r[i]]++;
	for(i=0;i<N;i++)
	{
		if(v[i] != 1 )
		{
			flag=1;
			printf("breaking at %d",i);
			break;
		}
	}
	if(flag==1)
		printf("\nroute is not valid");
	else
		printf("\nroute is valid");
}
/*Distance calculation of the initial solution */
long distH(float *px,float *py,long cit)
{
	float dx,dy;
	long cost=0;
	int i;
	for(i=0;i<(cit-1);i++)
	{
		dx=px[i]-px[i+1];
		dy=py[i]-py[i+1]; 
		cost+=sqrtf( (dx*dx) + (dy*dy) );
	}
	dx=px[i]-px[0];
	dy=py[i]-py[0]; 
	cost+=sqrtf( (dx*dx) + (dy*dy) );
	return cost;

}

int main(int argc, char *argv[])
{
	
	float *posx, *posy;
	float *px, *py,tm;
	char str[256];  
	float *d_posx, *d_posy;
	long x,y;
	int blk,thrd;
	clock_t start,end,start1,end1;
	long sol,tid,cities,no_pairs,dst,d;
	int *route,count=0;
	int ch, cnt, in1;
	float in2, in3;
        unsigned long long *d_dst_tid;
	FILE *f;

	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	px = (float *)malloc(sizeof(float) * cities);  if (px == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	py = (float *)malloc(sizeof(float) * cities);  if (py == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	route = (int *)malloc(sizeof(int) * cities);  if (route == NULL) {fprintf(stderr, "cannot allocate route\n");  exit(-1);}
	
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}
    	fflush(f);
	fclose(f);

	sol=cities*(cities-1)/2;
	int intl;	
	printf("\nChoose an initial solution setup approach\n1.Sequenced\n2.Random\n3.NN\n4.NI\n5.Greedy\n6.MST\n7.Christofide\n8.Clarke-Wright\n");
	scanf("%d",&intl);
	start = clock();
	switch(intl)
	{
		case 1:
			seq_init(route,cities);  
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities);
			dst=distH(px,py,cities);
			break;

		case 2:
			random_init(route,cities); 
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities); 
			dst=distH(px,py,cities);
			break;
		case 3:
			dst = nn_init(route,cities,posx,posy);
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities);
			break;
		case 4:
			nearest_insertion(route,posx,posy,cities);
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities);
			dst=distH(px,py,cities);
			break;
		case 5:
			greedy(route,posx,posy,cities);
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities);
			dst=distH(px,py,cities);
			break;
		case 6:
			mst_init(route,posx,posy,cities);
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities);
			dst=distH(px,py,cities);
			break;
		case 7:
			christofide_init(route, posx, posy, cities);
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities);
			dst=distH(px,py,cities);
			break;
		case 8:
			no_pairs = (cities-1)*(cities-2)/2;
			clarke_wright_init(route, posx, posy, cities, no_pairs);
			routeChecker(cities, route);
			setCoord(route,posx,posy,px,py,cities);
			dst=distH(px,py,cities);
			break;

	}
	end = clock();
	tm = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("\ninitial cost : %ld time : %f\n",dst,tm);

	start1 = clock();
	count = 1;
	unsigned long long dst_tid = (((long)dst+1) << 32) -1;
        unsigned long long dtid;
	long itr=floor(cities/2);
	int nx, ny;
	if(cities <= 32)
	{
		blk = 1 ;
		nx = cities;
		ny = cities;
	}
	else
	{
		blk = (cities - 1) / 32 + 1;
		nx = 32;
		ny = 32;
	}
	dim3 thrds (nx,ny);
	dim3 blks (blk,blk);
	if(hipSuccess!=hipMalloc((void**)&d_posx,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate x on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_posy,sizeof(float)*cities))
	printf("\nCan't allocate memory for coordinate y on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_dst_tid,sizeof(unsigned long long)))
	printf("\nCan't allocate memory for dst_tid on GPU");
    	if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
	printf("\nCan't transfer dst_tid on GPU");
	if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer px on GPU");
	if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
	printf("\nCan't transfer py on GPU");

	int strat;	
	printf("\n Choose a CUDA thread mapping strategy\n1.TPR\n2.TPRED\n3.TPRC\n4.TPN\n");
	scanf("%d",&strat);
	switch(strat)
	{
		case 1:

			if(cities<=1024)
			{
				blk=1;
				thrd=cities;
			}
			else
			{
				blk=(cities-1)/1024+1;
				thrd=1024;
			}
			
			tsp_tpr<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities);
			
			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");

			d = dtid >> 32;
			
			while( d < dst )
			{
				dst=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tpr<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities);
				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
		case 2:
			
			if(cities<1024)
			{
				blk=1;
				thrd=cities;
			}
			else
			{
				blk=(cities-1)/1024+1;
				thrd=1024;
			}	

			tsp_tpred<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities,itr);
			
			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");

			d = dtid >> 32;
			
			while( d < dst )
			{

				dst=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tpred<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities,itr);
				
				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
		case 3:
			
			tsp_tprc<<<blks,thrds>>>(d_posx,d_posy,dst,d_dst_tid,cities);
	
			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");
		  	d = dtid >> 32;
			
			while( d < dst )
			{
				dst=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tprc<<<blks,thrds>>>(d_posx,d_posy,dst,d_dst_tid,cities);
				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
		case 4:
			if(sol < 1024)
			{
				blk=1;
				thrd=sol;
			}
			else
			{
				blk=(sol-1)/1024+1;
				thrd=1024;
			}

			tsp_tpn<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities,sol);

			if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
			printf("\nCan't transfer minimal cost back to CPU");
			d = dtid >> 32;
			
			while( d < dst )
			{
				dst=d;
				tid = dtid & ((1ull<<32)-1); 
				x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
				y=tid-x*(cities-1)+(x*(x+1)/2)+1;
				twoOpt(x,y,px,py);
				if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer px on GPU");
				if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))
				printf("\nCan't transfer py on GPU");
				unsigned long long dst_tid = (((long)dst+1) << 32) -1;
				if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
				printf("\nCan't transfer dst_tid on GPU");

				tsp_tpn<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities,sol);

				if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
				printf("\nCan't transfer minimal cost back to CPU");
			  	d = dtid >> 32;
				count++;
			}
		break;
	}
	
	printf("\nMinimal Distance : %ld\n",d);

	printf("\nnumber of time climbed %d\n",count);
	end1 = clock();
	double t=((double) (end1 - start1)) / CLOCKS_PER_SEC;
	printf("\ntime : %f\n",t);
	hipFree(d_posy);
	hipFree(d_posx);
	hipFree(d_dst_tid);
	free(posx);
	free(posy);
	free(px);
	free(py);
	free(route);
	return 0;
}
